#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include ""  
  
#include <stdio.h>
#include <math.h>
#define Sphere_Num 100//���������200 cuda�˺��������
__constant__ int Sphere_Center_Device[Sphere_Num*3];
//__constant__ int Radius_Device [Sphere_Num];

__global__ void addKernel(int w,int h,int start,int end,int SphereNum,int * Radius_Device ,int * PictureBlock)  //��Ҫ��һ��cpp�ļ�������__device__��__global__����
{
	int x = blockIdx.x;				//w
	int y = blockIdx.y;				//h
	int z = threadIdx.x + start ;
	PictureBlock[threadIdx.x*w*h + y*w + x] = 0 ;

	for (int k = 0 ;k < SphereNum ; ++k)//��ÿ����     �жϣ�i��j��z���Ƿ�������    //�����ڴ�ѭ���ж������
 	{
		int xdistance2 = blockIdx.x - Sphere_Center_Device[k*3] ;
	    int ydistance2 = blockIdx.y - Sphere_Center_Device[k*3+1];
		int zdistance2 = threadIdx.x + start - Sphere_Center_Device[k*3+2];
		int tmp = ydistance2 * ydistance2 + xdistance2*xdistance2 +zdistance2* zdistance2;
		int  r = Radius_Device[k]*Radius_Device[k];
		if (tmp < r)
		{
			PictureBlock[threadIdx.x*w*h + y*w + x] = 255 ;
		}
	}
}





extern "C" 
hipError_t addWithCuda(int w,int h,int start,int end,int SphereNum,int* Sphere_Center,int *Radius_ ,int * PictureBlock)
{
	hipError_t cudaStatus;

	//int * Sphere_Center_Device = 0;
	int * Radius_Device		 = 0;
	int * Picture_Block_Device = 0;

	//cudaStatus = hipMalloc((void**)&Sphere_Center_Device, SphereNum*3*sizeof( int)     );  
	cudaStatus = hipMalloc((void**)&Radius_Device,        SphereNum*sizeof( int)       );
	cudaStatus = hipMalloc((void**)&Picture_Block_Device, (end-start+1)*w*h*sizeof(int));

	//hipMemcpy((void*)(Sphere_Center_Device), (void*)(Sphere_Center), SphereNum*3*sizeof( int), hipMemcpyHostToDevice);
	hipMemcpy((void*)( Radius_Device      ), (void*)(Radius_      ), SphereNum*sizeof( int)  , hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(Sphere_Center_Device),Sphere_Center,sizeof(int)*SphereNum*3);
	//hipMemcpyToSymbol(HIP_SYMBOL(Radius_Device       ),Radius_      ,sizeof(int)*SphereNum);


	if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMalloc failed! ");  
        goto Error;  
    } 

	dim3 blocks(w,h);
	dim3 threads(end-start+1);//RealPicNum
	addKernel<<<blocks,threads >>>(w,h,start,end,SphereNum,Radius_Device,Picture_Block_Device);



	hipMemcpy((void*)(PictureBlock), (void*)(Picture_Block_Device),(end-start+1)*w*h*sizeof(int), hipMemcpyDeviceToHost);

	cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));  
        goto Error;  
    }


Error:
	//hipFree(Sphere_Center_Device);
	hipFree(Radius_Device);
	hipFree(Picture_Block_Device);


	return cudaStatus;
}
